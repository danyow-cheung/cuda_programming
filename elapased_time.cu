hipEventRecord(start,0);
for(int i =0;i<2;++i){
    hipMemcpyAsync(inputDev+i*size ,inputHost+i*size ,size , hipMemcpyHostToDevice,stream[i] );
    MyKernel<<<100,512,0,,stream[i]>>>(outputDev+i*size,inputDev+i*size,size);
    hipMemcpyAsync(outputHost+i*size ,outputDev+i*size ,size , hipMemcpyHostToDevice,stream[i]);
}
hipEventRecord(stop,0);
hipEventSynchronize(stop);
float elapsed_time;
hipEventElapsedTime(&elapsed_time,start,stop);