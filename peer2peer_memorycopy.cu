hipSetDevice(0);
float* p0;
size_t size = 1024 * sizeof(float);
hipMalloc(&p0,size);
hipSetDevice(1);

float* p1;
hipMalloc(&p1,size);
hipSetDevice(0);
MyKernel<<<1000,128>>>(p0);
hipSetDevice(1);
hipMemcpyPeer(p1,1,p0,0,size);
MyKernel<<<1000,128>>>(p1);
