//3,2,8,7,3 Creating a Graph Using Stream Capture
hipGraph_t graph;
hipStreamBeginCapture(stream);

kernel_A<<< ..., stream >>>(...);
kernel_B<<< ..., stream >>>(...);
libraryCall(stream);
kernel_C<<< ..., stream >>>(...);

hipStreamEndCapture(stream, &graph);

// create the graph -it starts out empty 
hipGraphCreate(&graph,0);


// For the purpose of this example, we'll create
// the nodes separately from the dependencies to
// demonstrate that it can be done in two stages.
// Note that dependencies can also be specified
// at node creation.
hipGraphAddKernelNode(&a,graph,NULL,0,&nodeParams);
hipGraphAddKernelNode(&b,graph,NULL,0,&nodeParams);
hipGraphAddKernelNode(&c,graph,NULL,0,&nodeParams);
hipGraphAddKernelNode(&d,graph,NULL,0,&nodeParams);

// now set up dependenvvies on each node 
hipGraphAddDependencies(graph, &a, &b, 1);     // A->B
hipGraphAddDependencies(graph, &a, &c, 1);     // A->C
hipGraphAddDependencies(graph, &b, &d, 1);     // B->D
hipGraphAddDependencies(graph, &c, &d, 1);     // c->D

