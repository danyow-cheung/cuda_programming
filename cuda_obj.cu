hipGraph_t graph; // preexisting graph 
Object *object = new Object; // c++ object with possibly notrivial destructor 
hipUserObject_t cubObject;

hipUserObjectCreate(
    &cubObject, 
    object,  // use a cuda provided template wrapper for this api ,which supplies a callback to delete the c++ object pointer 
    1, // initiail refcount 
    hipUserObjectNoDestructorSync // acknowledge that the callback cannot be waited on vida CUDA
);

hipGraphRetainUserObject(
    graph,
    cubObject,
    1, // number of reference
    hipGraphUserObjectMove // transfer a reference owned by the caller // don't modify the total reference count
);


hipGraphRetainUserObject(
    graph,cubObject,1,hipGraphUserObjectMove //Transfer a reference owned by the caller (do
                             // not modify the total reference count)
);

//no more reference owned by this thread,no need to call release api
hipGraphExec_t graphExec;
hipGraphInstantiate(&graphExec,graph,nullptr,nullptr,0);

hipGraphDestroy(graph);
hipGraphLaunch(graphExec,0);
cudaGraphExecDestory(graphExec);

hipStreamSynchronize(0);
