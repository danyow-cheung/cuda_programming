#include "hip/hip_runtime.h"
__global__ void loopKernel(cudaGraphConditionalHandle handle){
    static int count = 10;
    cudaGraphSetConditional(handle,--count?1:0);
}

void graphSetup(){
    cudaGrapht_t graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t node;
    void *kernelArgs[1];

    hipGraphCreate(&graph,0);
    cudaGraphConditionalHandle  handle;
    cudaGraphConditionalHandleCreate(&handle,graph,1,cudaGraphCondAssignDefault);

    hipGraphNodeParams cParams = {hipGraphNodeTypeConditional};
    cParams.conditional.handle = handle;
    cParams.conditional.type = cudaGraphCondTypeWhile;
    cParams.conditional.size =1 ;
    hipGraphAddNode(&node,graph,NULL,0,&cParams);

    cudaGrapht_t bodyGraph = cParams.conditional.phGraph_out[0];
    hipGraphNodeParams params = {hipGraphNodeTypeKernel};
    params.kernel.func =  (void *)loopKernel;
    params.kernel.gridDim.x = params.kernel.gridDim.y = params.kernel.gridDim.z =1;
    params.kernel.blockDim.x = params.kernel.blockDim.y = params.kernel.blockDim.z =1;
    params.kernel.kernelParams = kernelArgs;
    kernelArgs[0] = &handle;

    hipGraphAddNode(&node,bodyGraph,NULL,0,&params);
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    hipGraphLaunch(graphExec, 0);
    hipDeviceSynchronize();

    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
}