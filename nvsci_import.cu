NvSciBufObj createNvSciBufObject(){
    //raw buffer attributes for cuda 
    NvSciBufType bufType = NvSciBufType_RawBuffer;
    uint64_t rawsize = SIZE;
    uint64_t align = 0 ;
    bool cpuaccess_flag = true;
    NvSciBufAttrValAccessPerm perm= NvSciBufAccessPerm_ReadWrite;

    NvSciRmGpuId gpuid[] ={};
    hipUUID uuid;
    hipDeviceGetUuid(&uuid, dev));

    memcpy(&gpuid[0].bytes, &uuid.bytes, sizeof(uuid.bytes));
    // Disable cache on dev
    NvSciBufAttrValGpuCache gpuCache[] = {{gpuid[0], false}};
    NvSciBufAttrValGpuCompression gpuCompression[] = {{gpuid[0], NvSciBufCompressionType_GenericCompressible}};
    // Fill in values
    NvSciBufAttrKeyValuePair rawbuffattrs[] = {
         { NvSciBufGeneralAttrKey_Types, &bufType, sizeof(bufType) },
         { NvSciBufRawBufferAttrKey_Size, &rawsize, sizeof(rawsize) },
         { NvSciBufRawBufferAttrKey_Align, &align, sizeof(align) },
         { NvSciBufGeneralAttrKey_NeedCpuAccess, &cpuaccess_flag, sizeof(cpuaccess_flag) },
         { NvSciBufGeneralAttrKey_RequiredPerm, &perm, sizeof(perm) },
         { NvSciBufGeneralAttrKey_GpuId, &gpuid, sizeof(gpuid) },
         { NvSciBufGeneralAttrKey_EnableGpuCache &gpuCache, sizeof(gpuCache) },
         { NvSciBufGeneralAttrKey_EnableGpuCompression &gpuCompression, sizeof(gpuCompression) }
    };

    // Create list by setting attributes
    err = NvSciBufAttrListSetAttrs(attrListBuffer, rawbuffattrs,
            sizeof(rawbuffattrs)/sizeof(NvSciBufAttrKeyValuePair));

    NvSciBufAttrListCreate(NvSciBufModule, &attrListBuffer);

    // Reconcile And Allocate
    NvSciBufAttrListReconcile(&attrListBuffer, 1, &attrListReconciledBuffer,
                       &attrListConflictBuffer)
    NvSciBufObjAlloc(attrListReconciledBuffer, &bufferObjRaw);
    return bufferObjRaw;
}

NvSciBufObj bufferObjRo; // Readonly NvSciBuf memory obj
// Create a duplicate handle to the same memory buffer with reduced permissions
NvSciBufObjDupWithReducePerm(bufferObjRaw, NvSciBufAccessPerm_Readonly, &bufferObjRo);
return bufferObjRo;


hipExternalMemory_t importNvSciBufObject(NvSciBufObj bufferObjRaw){
    //query nvscibuf object 
    NvSciBufAttrKeyValuePair bufattrs[] = {
        {NvSciBufRawBufferAttrKey_Size ,NULL,0},
        { NvSciBufGeneralAttrKey_GpuSwNeedCacheCoherency, NULL, 0 },
        { NvSciBufGeneralAttrKey_EnableGpuCompression, NULL, 0 }
    };
    NvSciBufAttrListGetAttrs(retList,bufattrs,sizeof(bufattrs)/sizeof(NvSciBufAttrKeyValuePair));
    ret_size = *(static_cast<const uint64_t*>(bufattrs[0].value));

    // Note cache and compression are per GPU attributes, so read values for specific gpu by comparing UUID
    // Read cacheability granted by NvSciBuf
    int numGpus = bufattrs[1].len / sizeof(NvSciBufAttrValueGpuCache);
    NvSciBufAttrValGpuCache[] cacheVal = (NvSciBufAttrValGpuCache *)bufattrs[1].value;
    bool ret_cacheVal;
    for(int i = 0;i<numGpus;i++){
        if (memcpy(gpuid[0].bytes,cacheVal[i].gpuId.bytes,sizeof(hipUUID)) ==0){
            ret_cacheVal = cacheVal[i].cacheabilit;
        }
    }

    //read compression granted by NvSciBuf
    numGpus = bufattrs[2].len / sizeof(NvSciBufAttrValGpuCompression);
    NvSciBufAttrValGpuCompression[] compVal = (NvSciBufAttrValGpuCompression *)bufattrs[2].value;
    NvSciBufCompressionType ret_compVal;
    for (int i = 0; i < numGpus; i++) {
        if (memcmp(gpuid[0].bytes, compVal[i].gpuId.bytes, sizeof(hipUUID)) == 0) {
            ret_compVal = compVal[i].compressionType);
        }
    }
    /*************** NvSciBuf Registration With CUDA **************/

    // Fill up hipExternalMemoryHandleDesc
    hipExternalMemoryHandleDesc memHandleDesc;
    memset(&memHandleDesc, 0, sizeof(memHandleDesc));
    memHandleDesc.type = cudaExternalMemoryHandleTypeNvSciBuf;
    memHandleDesc.handle.nvSciBufObject = bufferObjRaw;
    // Set the NvSciBuf object with required access permissions in this step
    memHandleDesc.handle.nvSciBufObject = bufferObjRo;
    memHandleDesc.size = ret_size;
    hipImportExternalMemory(&extMemBuffer, &memHandleDesc);
    return extMemBuffer;
    
}
