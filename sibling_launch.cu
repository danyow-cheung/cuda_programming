#include "hip/hip_runtime.h"
__global__ void launchSiblingGraph(hipGraphExec_t graph){
    hipGraphLaunch(graph,cudaStreamGraphFireAndForgetgetAsSibling);
}
void graphSetup(){
    hipGraphExec_t gExec1,gExec2;
    hipGraph_t g1,g2;

    // create instantiate and uploda the device graph 
    create_graph(&g2);
    hipGraphInstantiate(&gExec2,g2,hipGraphInstantiateFlagDeviceLaunch);
    hipGraphUpload(gExec2,stream);
    // create and instantiate the launching graph 
    hipStreamBeginCapture(stream,cudaStreamCaptureModelGlobal);
    launchSiblingGraph<<<1,1,0,stream>>>(gExec2);
    cudastreamEndCapture(stream,&g1);
    hipGraphInstantiate(&gExec1,g1);

    //launch the host graph ,which will in turn launch the device graph 
    hipGraphLaunch(gExec1,stream);

}