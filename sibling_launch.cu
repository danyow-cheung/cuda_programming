#include "hip/hip_runtime.h"
__global__ void launchSiblingGraph(hipGraphExec_t graph){
    hipGraphLaunch(graph,cudaStreamGraphFireAndForgetgetAsSibling);
}
void graphSetup(){
    hipGraphExec_t gExec1,gExec2;
    hipGraph_t g1,g2;

    // create instantiate and uploda the device graph 
    create_graph(&g2);
    
}