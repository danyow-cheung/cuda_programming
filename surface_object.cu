#include "hip/hip_runtime.h"
// simple copy kernel 
__global__ void copyKernel(hipSurfaceObject_t inputSurfObj,
                            hipSurfaceObject_t outputSurObj,
                            int width,int height)
{
    //calculate surface coordinations 
    unsigned int x=  blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y=  blockIdx.y * blockDim.y + threadIdx.y;
    if (x<width&&y<height){
        uchar4 data;
        //read from input surface
        surf2Dread(&data,inputSurfObj,x*4,y);
        //write to output surface 
        surf2DWrite(data,outputSurfObj,x*4,y);
    }
}

// host code 
int main(){
    const int height = 1024;
    const int width = 1024;
    //allocate and set some host data 
    unsigned char *h_data = (unsigned char *)std::malloc(sizeof(unsigned char)*width*height*4);
    for(int i=0;i<height*width*4;++i){
        h_data[i] = i;
    }

    //allocate cuda arrays  in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8,8,8,8,hipChannelFormatKindUnsigned);
    hipArray_t cuInputArray;
    hipMallocArray(&cuInputArray, &channelDesc, width, height,hipArraySurfaceLoadStore);

    hipArray_t cuOutputArray;
    hipMallocArray(&cuOutputArray,&channelDesc,width,height,hipArraySurfaceLoadStore);

    //set pitch of the source (the width in memory in bytes of the 2d array )
    // pointed to by src,including padding ,we don't have any padding 
    const size_t spitch = 4*width*sizeof(unsigned char);
    //copy data located at address h_data in host memory to device memory 
    cudaMemcpy2DTOArray(cuInputArray,0,0,h_data,spitch,4*width*sizeof(unsigned char),height,hipMemcpyHostToDevice);

    //specify surface 
    struct hipResourceDesc resDesc;
    memst(&resDesc,0,sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    //create the surface objects
    resDesc.res.array.array = cuInputArray;
    hipSurfaceObject_t inputSurfObj = 0;
    hipCreateSurfaceObject(&inputSurObj,&resDesc);
    resDesc.res.array.array = cuOutputArray;
    hipSurfaceObject_t outputSurfObj = 0;
    hipCreateSurfaceObject(&outputSurfObj,&resDesc);

    //invoke kernel 
    dim3 threadsperBlock(16,16);
    dim3 numBlocks((width + threadsperBlock.x - 1) / threadsperBlock.x,
                    (height + threadsperBlock.y - 1) / threadsperBlock.y);
    copyKernel<<<numBlocks, threadsperBlock>>>(inputSurfObj, outputSurfObj, width,
                                                height);
    //copy data from device back to host 
    cudaMemecpy2DFromArray(h_data,spitch,cuOutputArray,0,0,4*width*sizeof(unsigned char),height,hipMemcpyDeviceToHost);

    //destropy surface objects 
    hipDestroySurfaceObject(inputSurfObj);
    hipDestroySurfaceObject(outputSurObj);

    //free device memory 
    hipFreeArray(cuInputArray);
    hipFreeArray(cuOutputArray);
    //free host memory
    free(h_data);
    return 0 ;
}