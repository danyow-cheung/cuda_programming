#include "hip/hip_runtime.h"
__global__ void launchTailGraph(hipGraphExec_t graph){
    hipGraphLaunch(graph,cudaStreamGraphTailLaunch);
}
void graphSetup(){
    hipGraphExec_t gExec1,gExec2;
    hipGraph_t g1,g2;
    // create instantiate and upload the device graph 
    create_graph(&g2);
    hipGraphInstantiate(&gExec2,g2,hipGraphInstantiateFlagDeviceLaunch);
    hipGraphUpload(gExec2,stream);

    // create and instantiate the launching graph 
    hipStreamBeginCapture(stream,cudaStreamCaptureModelGlobal);
    launchTailGraph<<<1,1,0,stream>>>(gExec2);
    hipStreamEndCapture(stream,&g1);
    hipGraphInstantiate(&gExec1,g1);
    // launch the host graph which will in turn launch the device graph 
    hipGraphLaunch(gExec1,stream);
    
}